#include "hip/hip_runtime.h"
#include <iostream>


__device__ void calculate(int * p_evaluations){
    int blockId = blockIdx.x;

    p_evaluations[blockId]++;

}

__global__ void update(int * p_evaluations){

    calculate(p_evaluations);
    calculate(p_evaluations);  

}


int main(){

    int p = 5;

    thurst::device_vector<int> d_evaluations(p);

    int * p_evaluations = thrust::raw_ponter_cast(&d_evaluations[0]);

    update<<< p, 1 >>> (p_evaluations);
    hipDeviceSynchronize();

    int sum = thrust::reduce(d_evaluations.begin(), d_evaluations.end(), 0, thrust::plus<int>());

    
    printf("%d\n", sum);
    


}