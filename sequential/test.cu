
#include <hip/hip_runtime.h>
#include <iostream>


__global__ void update(int * e){

    (*e)++;

}


int main(){

    int * e;
    
    hipMallocManaged(&e, sizeof(int));
    
    (*e) = 0;
    (*e)++;

    //printf("AAA\n");

    update<<< 10, 5 >>> (e);
    hipDeviceSynchronize();

    printf("%d\n", (*e));
    


}