
#include "util.cuh"
#include "nelderMead.cuh"


int main() {

    NelderMead parameters;
    parameters.benchmark_problem = NONE;
    parameters.problem_type = NO_PROBLEM;
    
    ABOffLattice * parametersAB;

    std::ifstream input_file("resources/inputs/input.txt");

    if(!readInput(parameters, input_file, parametersAB)){
        return 1;
    }

    printf("-------------------- PARAMETERS --------------------\n");
    printf("Executions: %d\n", parameters.executions_number);
    printf("Iterations: %d\n", parameters.iterations_number);
    printf("Dimension:  %d\n", parameters.dimension);
    printf("----------------------------------------------------\n");

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsed_time;

    if(parameters.problem_type == BENCHMARK){

        std::vector<NelderMeadResult> results(parameters.executions_number);
        
        for(int i = 0; i < parameters.executions_number; i++){
            parameters.p_start = &parameters.starting_points[i][0];

            hipEventRecord(start);

            results[i] = nelderMead(parameters);
            
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed_time, start, stop);

            printf(" - Execution %d:\n", i + 1);

            printf("Best: %.7f\n", results[i].best);
            
            if(parameters.show_best_vertex){
                printf("Best Vertex:\n");

                for(int j = 0; j < parameters.dimension; j++){
                    printf("%.7f ", results[i].best_vertex[j]);
                }
                printf("\n");
            }

            printf("Evaluations: %d\n", results[i].evaluations_used);
            printf("Elapsed Time: %.7f\n", elapsed_time / 1000.0);
        }

        float mean = 0.0f;
        for(int i = 0; i < parameters.executions_number; i++){
            mean += results[i].best;
        }
        mean /= parameters.executions_number;

        printf("\nMean of Best vertexes: %.7f\n", mean);

    }else if(parameters.problem_type == AB_OFF_LATTICE){

        ABOffLattice * d_parametersAB;

        hipMalloc(&d_parametersAB, sizeof(ABOffLattice));
        hipMemcpy(d_parametersAB, parametersAB, sizeof(ABOffLattice), hipMemcpyHostToDevice);

        char aa_sequence[150];
        memset(aa_sequence, 0, sizeof(char) * 150);
        strcpy(aa_sequence, (*parametersAB).aminoacid_sequence);
        hipMemcpyToSymbol(HIP_SYMBOL(aminoacid_sequence), (void *) aa_sequence, 150 * sizeof(char));

        hipEventRecord(start);

        NelderMeadResult result = nelderMead(parameters, (void*) parametersAB, (void*) d_parametersAB );

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);
        
        printf("Best: %.7f\n", result.best);


        if(parameters.show_best_vertex){
            printf("Best Vertex:\n");
            
            for(int i = 0; i < parameters.dimension; i++){
                printf("%.7f ", result.best_vertex[i]);
            }
        }
        printf("\nEvaluations: %d\n", result.evaluations_used);
        printf("Elapsed Time: %.7f\n", elapsed_time / 1000.0);

    }

}

